
#include <hip/hip_runtime.h>
#include <iostream>
#include <omp.h>

int main(){
    int particles = 2416;

    double *acc, *aacc, *vel, *pos, *m, *distances; 
    double *dacc, *daacc, *dvel, *dpos, *dm, *ddistances; 

    acc   = (double*)malloc(sizeof(double) * particles*3);
    aacc   = (double*)malloc(sizeof(double) * particles*3);
    vel = (double*)malloc(sizeof(double) * particles*3);
    pos = (double*)malloc(sizeof(double) * particles*3);
    m = (double*)malloc(sizeof(double) * particles);
    distances = (double*)malloc(sizeof(double) * int((particles*(particles-1))*0.5));

    hipMalloc((void **) &dacc, sizeof(double) * particles*3);
    hipMalloc((void **) &daacc, sizeof(double) * particles*3);
    hipMalloc((void **) &dvel, sizeof(double) * particles*3);
    hipMalloc((void **) &dpos, sizeof(double) * particles*3);
    hipMalloc((void **) &dm, sizeof(double) * particles);
    hipMalloc((void **) &ddistances, sizeof(double) * int((particles*(particles-1))*0.5));

    for (int i = 0; i < int((particles*(particles-1))*0.5); i++)
    {
        if (i < particles*3)
        {
            acc[i] = i*0.5;
            aacc[i] = i*0.5;
            vel[i] = i*0.5;
            pos[i] = i*0.5;
        }
        if (i < particles)
        {
            m[i] = i*1.5;
        }
        distances[i] = i*2.5;
    }


    double time0 = omp_get_wtime(); 
    hipMemcpy(dacc,acc,sizeof(double)*particles*3,hipMemcpyHostToDevice);
    hipMemcpy(daacc,aacc,sizeof(double)*particles*3,hipMemcpyHostToDevice);
    hipMemcpy(dvel,vel,sizeof(double)*particles*3,hipMemcpyHostToDevice);
    hipMemcpy(dpos,pos,sizeof(double)*particles*3,hipMemcpyHostToDevice);
    hipMemcpy(dm,m,sizeof(double)*particles,hipMemcpyHostToDevice);
    hipMemcpy(ddistances,distances,sizeof(double)*int((particles*(particles-1))*0.5),hipMemcpyHostToDevice);

    hipMemcpy(acc,dacc,sizeof(double)*particles*3,hipMemcpyDeviceToHost);
    hipMemcpy(aacc,daacc,sizeof(double)*particles*3,hipMemcpyDeviceToHost);

    double time1 = omp_get_wtime();

    hipFree(dacc);
    hipFree(daacc);
    hipFree(dpos);
    hipFree(dvel);
    hipFree(dm);
    hipFree(ddistances);

    delete [] acc;
    delete [] aacc;
    delete [] vel;
    delete [] pos;
    delete [] m;
    delete [] distances;


    std::cout<<"Equivalent copying took "<<(time1-time0)*994<<" seconds"<<std::endl;


}